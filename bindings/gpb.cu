/*
 * Written by Andreas Mueller
 * Basically copy&paste from damascene.cu
 * Provides C interface to highlevel gPB
 *
 * Compute gPB operator using damascene with cuda
 *
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <fcntl.h>
#include <float.h>
#include <unistd.h>
#include "texton.h"
#include "convert.h"
#include "intervening.h"
#include "lanczos.h"
#include "stencilMVM.h"

#include "localcues.h"
#include "combine.h"
#include "nonmax.h"
#include "spectralPb.h"
#include "globalPb.h"
#include "skeleton.h"

#include <iostream>
#define TEXTON64 2
#define TEXTON32 1

void transpose(int width, int height, float* input, float* output) {
  for(int row = 0; row < height; row++) {
    for(int col = 0; col < width; col++) {
      output[col * height + row] = input[row * width + col];
    }
  }                                         
}

//void gpb(const unsigned int* in_image,unsigned int width, unsigned int height, float* borders,int* textons, float* orientations, int device_num)
void gpb(const float* in_image,unsigned int width, unsigned int height, float* borders, int* textons, float* orientations, int device_num)
{
	hipInit(0);
	hipSetDevice(device_num);
/*
	uint* devRgbU;
	// copy in_image to device:

	uint nPixels = width * height;
	hipMalloc((void**)&devRgbU, nPixels*sizeof(uint));
	size_t totalMemory, availableMemory;
	hipMemGetInfo(&availableMemory,&totalMemory );
	std::cout<<"Available " << availableMemory << "out of "<< totalMemory<<  " bytes on GPU" <<std::endl;
	hipMemcpy(devRgbU, in_image, nPixels*sizeof(uint), hipMemcpyHostToDevice);

	float* devGreyscale;
	rgbUtoGreyF(width, height, devRgbU, &devGreyscale);

	int nTextonChoice = TEXTON64;

	int* devTextons;
	findTextons(width, height, devGreyscale, &devTextons, nTextonChoice);
	//int* hostTextons = (int*)malloc(sizeof(int)*width*height); 
	hipMemcpy(textons, devTextons, sizeof(int)*width*height, hipMemcpyDeviceToHost); 


	float* devL;
	float* devA;
	float* devB;
	rgbUtoLab3F(width, height, 2.5, devRgbU, &devL, &devA, &devB);
	normalizeLab(width, height, devL, devA, devB);

	int border = 30;
	float* devLMirrored;
	mirrorImage(width, height, border, devL, &devLMirrored);
	hipDeviceSynchronize();
	hipFree(devRgbU);
	hipFree(devGreyscale);
	
	float* devBg;
	float* devCga;
	float* devCgb;
	float* devTg;
	int matrixPitchInFloats;

	localCues(width, height, devL, devA, devB, devTextons, &devBg, &devCga, &devCgb, &devTg, &matrixPitchInFloats, nTextonChoice);
	hipFree(devTextons);
	hipFree(devL);
	hipFree(devA);
	hipFree(devB);
	float* devMPbO;
	float *devCombinedGradient;
	combine(width, height, matrixPitchInFloats, devBg, devCga, devCgb, devTg, &devMPbO, &devCombinedGradient, nTextonChoice);
	CUDA_SAFE_CALL(hipFree(devBg));
	CUDA_SAFE_CALL(hipFree(devCga));
	CUDA_SAFE_CALL(hipFree(devCgb));
	CUDA_SAFE_CALL(hipFree(devTg));
*/
	float* devMPb;
	devMPb = (float*)in_image;

	int nPixels = width * height;

	int matrixPitchInFloats = findPitchInFloats(nPixels);

	hipMalloc((void**)&devMPb, sizeof(float) * nPixels);
//	nonMaxSuppression(width, height, devMPbO, matrixPitchInFloats, devMPb);
	//int devMatrixPitch = matrixPitchInFloats * sizeof(float);
	int radius = 5;
	//int radius = 10;

	Stencil theStencil(radius, width, height, matrixPitchInFloats);
	int nDimension = theStencil.getStencilArea();
	float* devMatrix;
	intervene(theStencil, devMPb, &devMatrix);
	printf("Intervening contour completed\n");

	float* eigenvalues;
	float* devEigenvectors;
	//int nEigNum = 17;
	int nEigNum = 9;
	float fEigTolerance = 1e-3;
	generalizedEigensolve(theStencil, devMatrix, matrixPitchInFloats, nEigNum, &eigenvalues, &devEigenvectors, fEigTolerance);

	printf("Generalized Eigenvectors solve completed\n");

	float* devSPb = 0;
	size_t devSPb_pitch = 0;
	CUDA_SAFE_CALL(hipMallocPitch((void**)&devSPb, &devSPb_pitch, nPixels *  sizeof(float), 8));
	hipMemset(devSPb, 0, matrixPitchInFloats * sizeof(float) * 8);

	spectralPb(eigenvalues, devEigenvectors, width, height, nEigNum, devSPb, matrixPitchInFloats);
/* ???
	float* devGPb = 0;
	CUDA_SAFE_CALL(hipMalloc((void**)&devGPb, sizeof(float) * nPixels));
	float* devGPball = 0;
	CUDA_SAFE_CALL(hipMalloc((void**)&devGPball, sizeof(float) * matrixPitchInFloats * 8));
 	StartCalcGPb(nPixels, matrixPitchInFloats, 8, devCombinedGradient, devSPb, devMPb, devGPball, devGPb);
	float* devGPb_thin = 0;
	CUDA_SAFE_CALL(hipMalloc((void**)&devGPb_thin, nPixels * sizeof(float) ));
	PostProcess(width, height, width, devGPb, devMPb, devGPb_thin); //note: 3rd param width is the actual pitch of the image
	NormalizeGpbAll(nPixels, 8, matrixPitchInFloats, devGPball);
*/
	hipDeviceSynchronize();
	printf("CUDA Status : %s\n", hipGetErrorString(hipGetLastError()));
	
	//float* hostGPb = (float*)malloc(sizeof(float)*nPixels);
	//memset(hostGPb, 0, sizeof(float) * nPixels);
	std::cout << "nPixels: " << nPixels << std::endl;
/* ???
	hipMemcpy(borders, devGPb, sizeof(float)*nPixels, hipMemcpyDeviceToHost); //TODO: put in again
*/
	/*hipMemcpy(out_image, devGreyscale, sizeof(float)*nPixels, hipMemcpyDeviceToHost);*/
	//cutSavePGMf(outputPGMfilename, hostGPb, width, height);
	//writeFile(outputPBfilename, width, height, hostGPb);

	/* thin image */
	//float* hostGPb_thin = (float*)malloc(sizeof(float)*nPixels);
	//memset(hostGPb_thin, 0, sizeof(float) * nPixels);
	/*hipMemcpy(hostGPb_thin, devGPb_thin, sizeof(float)*nPixels, hipMemcpyDeviceToHost);*/

	//cutSavePGMf(outputthinPGMfilename, hostGPb_thin, width, height);
	//writeFile(outputthinPBfilename, width, height, hostGPb);
	//free(hostGPb_thin);
	/* end thin image */
/* ???
  float* hostGPbAll = (float*)malloc(sizeof(float) * matrixPitchInFloats * 8);
  hipMemcpy(hostGPbAll, devGPball, sizeof(float) * matrixPitchInFloats * 8, hipMemcpyDeviceToHost);
  //int oriMap[] = {0, 1, 2, 3, 4, 5, 6, 7};
  //int oriMap[] = {4, 5, 6, 7, 0, 1, 2, 3};
  int oriMap[] = {3, 2, 1, 0, 7, 6, 5, 4};
  for(int i = 0; i < 8; i++) {
    transpose(width, height, hostGPbAll + matrixPitchInFloats * oriMap[i], orientations + width * height * i);
  }
*/
  //int dim[3];
  //dim[0] = 8; 
  //dim[1] = width;
  //dim[2] = height;
  //writeArray(outputgpbAllfilename, 3, dim, hostGPbAllConcat);
  
  
  //for(int orientation = 0; orientation < 8; orientation++) {
	//sprintf(nIndicator, "_%i_Pb.pgm", orientation);
	//cutSavePGMf("orientation.pgm", hostGPbAll + matrixPitchInFloats * orientation, width, height);
  //}
  
	/*free(hostGPb);*/
// ???	free(hostGPbAll);
	//free(hostGPbAllConcat);


	CUDA_SAFE_CALL(hipFree(devEigenvectors));
// ???	CUDA_SAFE_CALL(hipFree(devCombinedGradient));
	CUDA_SAFE_CALL(hipFree(devSPb));
// ???	CUDA_SAFE_CALL(hipFree(devGPb));
// ???	CUDA_SAFE_CALL(hipFree(devGPb_thin));
// ???	CUDA_SAFE_CALL(hipFree(devGPball));
	hipDeviceReset();
}

int main(){

	return 0;
}

